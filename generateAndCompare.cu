﻿#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define ADDRESS_LENGTH 20
#define PRIVATE_KEY_LENGTH 32

// Custom comparison function for device code
__device__ int compareBytes(const unsigned char* a, const unsigned char* b, int length)
{
    for (int i = 0; i < length; i++)
    {
        if (a[i] < b[i]) return -1;
        if (a[i] > b[i]) return 1;
    }
    return 0;
}

// Binary search on sorted address array
__device__ int binarySearch(const unsigned char* sortedData, int dataSize, const unsigned char* target)
{
    int left = 0;
    int right = dataSize - 1;
    while (left <= right)
    {
        int mid = left + (right - left) / 2;
        int cmp = compareBytes(sortedData + mid * ADDRESS_LENGTH, target, ADDRESS_LENGTH);
        if (cmp == 0) return mid;
        if (cmp < 0) left = mid + 1;
        else right = mid - 1;
    }
    return -1;
}

extern "C" __global__ void generateAndCompare(
    unsigned char* addresses,
    unsigned char* privateKeys, // Added to store private keys
    unsigned char** sortedFiles, int* fileSizes,
    unsigned int* foundCount, unsigned char* foundAddresses,
    unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed + idx, 0, 0, &state);

    // Generate private key (32 bytes)
    unsigned char privateKey[PRIVATE_KEY_LENGTH];
    for (int i = 0; i < PRIVATE_KEY_LENGTH; i++)
        privateKey[i] = (unsigned char)(hiprand(&state) & 0xFF);

    // Store private key in output buffer
    int privOffset = idx * PRIVATE_KEY_LENGTH;
    memcpy(privateKeys + privOffset, privateKey, PRIVATE_KEY_LENGTH);

    // Dummy address derivation (for comparison only)
    unsigned char address[ADDRESS_LENGTH];
    for (int i = 0; i < ADDRESS_LENGTH; i++)
        address[i] = privateKey[i + 12]; // Simplified; host will recompute correctly

    // Copy to output buffer
    int addrOffset = idx * ADDRESS_LENGTH;
    memcpy(addresses + addrOffset, address, ADDRESS_LENGTH);

    // Compare with sorted files
    int fileIdx = address[0];
    if (fileSizes[fileIdx] > 0)
    {
        int found = binarySearch(sortedFiles[fileIdx], fileSizes[fileIdx], address);
        if (found >= 0)
        {
            unsigned int pos = atomicAdd(foundCount, 1);
            memcpy(foundAddresses + pos * ADDRESS_LENGTH, address, ADDRESS_LENGTH);
        }
    }
}